#include "hip/hip_runtime.h"
/*
	Noel Lopes is an Assistant Professor at the Polytechnic Institute of Guarda, Portugal
	Copyright (C) 2009, 2010, 2011, 2012 Noel de Jesus Mendon�a Lopes

	This file is part of GPUMLib.

	GPUMLib is free software: you can redistribute it and/or modify
	it under the terms of the GNU General Public License as published by
	the Free Software Foundation, either version 3 of the License, or
	(at your option) any later version.

	This program is distributed in the hope that it will be useful,
	but WITHOUT ANY WARRANTY; without even the implied warranty of
	MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
	GNU General Public License for more details.

	You should have received a copy of the GNU General Public License
	along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "MBPkernels.h"

#define OUTPUT_NEURON threadIdx.x
#define OUTPUT_INCLUDING_BIAS (threadIdx.x + 1)
#define NUM_OUTPUTS blockDim.x

#define NEURON threadIdx.y
#define NUM_NEURONS blockDim.y

#define NUM_INPUTS_OUTPUT_NEURON (NUM_NEURONS + 1)

#define PATTERN blockIdx.x

namespace GPUMLib {

KERNEL CalculateLocalGradient(cudafloat * rmsF, cudafloat * bestRMS, cudafloat maxErrorGrowth, cudafloat * outputs, cudafloat * weights, cudafloat * m, int mOffset, int totalNeuronsWithSelectiveActivation, cudafloat * localGradientNextLayer, cudafloat * localGradient, cudafloat * localGradientSpaceNet) {
	extern __shared__ cudafloat lg[];
	
	if (bestRMS != nullptr) {
		__shared__ cudafloat rms;
		__shared__ cudafloat bRMS;
		
		rms = *rmsF;
		bRMS = *bestRMS;
		if (rms >= bRMS * maxErrorGrowth) return;
	}

	cudafloat * lgNextLayer = (lg + (NUM_OUTPUTS * NUM_NEURONS));
			
	if (NEURON == 0) lgNextLayer[OUTPUT_NEURON] = localGradientNextLayer[PATTERN * NUM_OUTPUTS + OUTPUT_NEURON];
	
	int connection = OUTPUT_NEURON * NUM_INPUTS_OUTPUT_NEURON + NEURON + 1;    
	int threadId = (NEURON * NUM_OUTPUTS + OUTPUT_NEURON);
	
	__syncthreads();    
	
	lg[threadId] = weights[connection] * lgNextLayer[OUTPUT_NEURON];
	__syncthreads();

	int numberElemSum = NUM_OUTPUTS;
	for(int sumUpTo = (numberElemSum >> 1); numberElemSum > 1; sumUpTo = (numberElemSum >> 1)) {
		int nextNumberElemSum = sumUpTo;
		if (numberElemSum & 1) nextNumberElemSum++;
	
		if (OUTPUT_NEURON < sumUpTo) lg[threadId] += lg[threadId + nextNumberElemSum];
		
		numberElemSum = nextNumberElemSum;
		
		__syncthreads();
	}
	
	if (OUTPUT_NEURON == 0) {
		int n = PATTERN * NUM_NEURONS + NEURON;
		
		cudafloat Fh = outputs[n];
	
		cudafloat lgn = lg[threadId];
	
		if (m != nullptr) {
			int nSelAct = PATTERN * totalNeuronsWithSelectiveActivation + NEURON + mOffset;

			cudafloat M = m[nSelAct];
			if (M == CUDA_VALUE(0.0)) {
				localGradientSpaceNet[nSelAct] = CUDA_VALUE(0.0);
			} else {
				Fh = Fh / M;
				localGradientSpaceNet[nSelAct] = lgn * Fh * CUDA_SIGMOID_DERIVATE(M);
			}
			lgn *= M;
		}
	
		localGradient[n] = lgn * CUDA_SIGMOID_DERIVATE(Fh);
	}
}

}