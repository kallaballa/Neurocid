/*
	Noel Lopes is an Assistant Professor at the Polytechnic Institute of Guarda, Portugal
	Copyright (C) 2009, 2010, 2011, 2012 Noel de Jesus Mendon�a Lopes

	This file is part of GPUMLib.

	GPUMLib is free software: you can redistribute it and/or modify
	it under the terms of the GNU General Public License as published by
	the Free Software Foundation, either version 3 of the License, or
	(at your option) any later version.

	This program is distributed in the hope that it will be useful,
	but WITHOUT ANY WARRANTY; without even the implied warranty of
	MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
	GNU General Public License for more details.

	You should have received a copy of the GNU General Public License
	along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <cstdlib>
#include "random.h"

using namespace std;

namespace GPUMLib {

hiprandGenerator_t Random::randomGenerator = nullptr;
hiprandRngType_t Random::randomGeneratorType = HIPRAND_RNG_PSEUDO_DEFAULT;

hiprandGenerator_t Random::RandomGenerator() {
	if (randomGenerator == nullptr) {
		hiprandCreateGenerator(&randomGenerator, randomGeneratorType);
		atexit(&CleanUp);
	}

	return randomGenerator;
}

void Random::CleanUp() {
	if (randomGenerator != nullptr) {
		hiprandDestroyGenerator(randomGenerator);
		randomGenerator = nullptr;
	}
}

void Random::SetSeed(unsigned long long seed, hiprandRngType_t generatorType) {
	if (generatorType != randomGeneratorType) {
		randomGeneratorType = generatorType;
		CleanUp();		
	}

	hiprandSetPseudoRandomGeneratorSeed(RandomGenerator(), seed);
}

void Random::Fill(DeviceArray<float> & a) {
	hiprandGenerateUniform(RandomGenerator(), a.Pointer(), a.Length());
}

}